#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <random>
#include "kernels.cuh"



int main()
{
	unsigned int n = 1000*256*256;
	float *h_prod;
	float *d_prod;
	float *h_x, *h_y;
	float *d_x, *d_y;

	// allocate memory
	h_prod = (float*)malloc(sizeof(float)); 
	h_x = (float*)malloc(n*sizeof(float));
	h_y = (float*)malloc(n*sizeof(float));
	hipMalloc((void**)&d_prod, sizeof(float));
	hipMalloc((void**)&d_x, n*sizeof(float));
	hipMalloc((void**)&d_y, n*sizeof(float));
	hipMemset(d_prod, 0.0, sizeof(float));

	// fill host array with data
	for(unsigned int i=0;i<n;i++){
		h_x[i] = float(rand()%n) / n;
		h_y[i] = float(rand()%n) / n;
	}


	// timing variables
	float gpu_elapsed_time = 0.0;
	hipEvent_t gpu_start, gpu_stop;
	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);


	// copy data to device
	hipMemcpy(d_x, h_x, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, n*sizeof(float), hipMemcpyHostToDevice);
	hipEventRecord(gpu_start, 0);


	// call dot product kernel
	dim3 gridSize = 256;
	dim3 blockSize = 256;
	dot_product_kernel<<<gridSize, blockSize>>>(d_x, d_y, d_prod, n);

	// copy data back to host
	hipEventRecord(gpu_stop, 0);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);
	hipMemcpy(h_prod, d_prod, sizeof(float), hipMemcpyDeviceToHost);
	

	// report results
	std::cout<<"dot product computed on GPU is: "<<*h_prod<<" and took "<<gpu_elapsed_time<<std::endl;


	// run CPU based dot product to compare times to GPU code
	clock_t cpu_start = clock();
	double temp = 0.0;
	for(unsigned int i=0;i<n;i++){
		temp += h_x[i]*h_y[i];
	}
	//*h_prod = temp;
	clock_t cpu_stop = clock();
	clock_t cpu_elapsed_time = 1000*(cpu_stop - cpu_start)/CLOCKS_PER_SEC;
	std::cout<<"dot product computed on CPU is: "<<temp<<" and took "<<cpu_elapsed_time<<std::endl;


	// free memory
	free(h_prod);
	free(h_x);
	free(h_y);
	hipFree(d_prod);
	hipFree(d_x);
	hipFree(d_y);

}
